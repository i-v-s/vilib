#include "hip/hip_runtime.h"
/*
 * Functions for creating image pyramids on the GPU
 * pyramid_gpu.cu
 *
 * Copyright (C) 2019-2020 Balazs Nagy,
 * Robotics and Perception Group, University of Zurich
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <opencv2/imgproc.hpp>
#include "vilib/preprocess/pyramid.h"
#include "vilib/cuda_common.h"

namespace vilib {
#define USE_TEXTURE_MEMORY  0

#define USE_TEXTURE_OBJECTS 0

#if (USE_TEXTURE_MEMORY == 1) && (USE_TEXTURE_OBJECTS == 0)
static texture<unsigned char, hipTextureType2D, hipReadModeNormalizedFloat> d_image_in_tex;
#endif /* USE_TEXTURE_MEMORY */

#if USE_TEXTURE_MEMORY
template <typename T, const int N>
static __global__ void image_halfsample_gpu_tex_kernel(T * __restrict__ d_image_out,
#if USE_TEXTURE_OBJECTS
                                                       hipTextureObject_t d_image_in_tex,
#endif /* USE_TEXTURE_OBJECTS */
                                                       const unsigned int width_dst_px,
                                                       const unsigned int height_dst_px,
                                                       const unsigned int pitch_dst_px) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if((x < width_dst_px) && (y < height_dst_px)) {
    const int dst = y*pitch_dst_px/N + x; //every thread writes N bytes. the next row starts at pitch_Dst_px/N
    float src_x = N*2.0f*x + 1.0f;        //every thread reads in 2x4 bytes
    float src_y = 2.0f*y + 1.0f;
    #pragma unroll N
    for(int i=0;i<N;++i,src_x += 2.0f) {
      *(((unsigned char*)(d_image_out+dst))+i) = (unsigned char)(255.0f*
#if USE_TEXTURE_OBJECTS
      tex2D<float>
#else
      tex2D
#endif /* USE_TEXTURE_OBJECTS */
        (d_image_in_tex,src_x,src_y));
    }
  }
}
#else
template <typename T, const int N>
__global__ void image_halfsample_gpu_kernel(const uchar2 * __restrict__ d_image_in,
                                 const unsigned int pitch_src_px,
                                 T * __restrict__ d_image_out,
                                 const unsigned int width_px,
                                 const unsigned int height_px,
                                 const unsigned int pitch_dst_px) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if((x < width_px) && (y < height_px)) {
    const int dst  = y*pitch_dst_px/N + x; //every thread writes N bytes. the next row starts at pitch_dst_px/N
    int src_top    = y*pitch_src_px + x*N; //every thread reads in Nx2 bytes
    int src_bottom = y*pitch_src_px + x*N + (pitch_src_px/2);
    #pragma unroll N
    for(int i=0;i<N;++i) {
      const uchar2 t2 = d_image_in[src_top++];
      const uchar2 b2 = d_image_in[src_bottom++];
      *(((unsigned char*)(d_image_out+dst))+i) = (unsigned char)(((unsigned int)t2.x + (unsigned int)t2.y + (unsigned int)b2.x + (unsigned int)b2.y)>>2);
    }
  }
}
#endif /* USE_TEXTURE_MEMORY */

static inline __host__ void pyramid_create_level_gpu(const unsigned char * d_img_src,
                                                      unsigned char * d_img_dst,
                                                      std::size_t & img_src_pitch,
                                                      std::size_t & img_dst_pitch,
                                                      std::size_t & img_src_width,
                                                      std::size_t & img_dst_width,
                                                      std::size_t & img_src_height,
                                                      std::size_t & img_dst_height,
                                                      hipStream_t stream) {
#if USE_TEXTURE_MEMORY
#if USE_TEXTURE_OBJECTS
  hipTextureObject_t tex_object;
  hipResourceDesc tex_res;
  memset(&tex_res,0,sizeof(hipResourceDesc));
  tex_res.resType = hipResourceTypePitch2D;
  tex_res.res.pitch2D.width = img_src_width;
  tex_res.res.pitch2D.height = img_src_height;
  tex_res.res.pitch2D.pitchInBytes = img_src_pitch;
  tex_res.res.pitch2D.devPtr = (void*)d_img_src;
  tex_res.res.pitch2D.desc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);

  hipTextureDesc tex_desc;
  memset(&tex_desc,0,sizeof(hipTextureDesc));
  tex_desc.normalizedCoords = 0;
  tex_desc.filterMode = hipFilterModeLinear;
  tex_desc.addressMode[0] = hipAddressModeClamp;
  tex_desc.addressMode[1] = hipAddressModeClamp;
  tex_desc.readMode = hipReadModeNormalizedFloat;

  // create texture object on the fly
  CUDA_API_CALL(hipCreateTextureObject(&tex_object,&tex_res,&tex_desc,NULL));
#else
  // Bind the texture memory
  d_image_in_tex.addressMode[0] = hipAddressModeClamp;
  d_image_in_tex.addressMode[1] = hipAddressModeClamp;
  d_image_in_tex.filterMode     = hipFilterModeLinear;
  d_image_in_tex.normalized     = 0;
  d_image_in_tex.channelDesc    = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);
  CUDA_API_CALL(hipBindTexture2D(NULL,&d_image_in_tex,d_img_src,&d_image_in_tex.channelDesc,img_src_width,img_src_height,img_src_pitch));
#endif /* USE_TEXTURE_OBJECTS */
#else
  // Unused parameters
  (void)img_src_width;
  (void)img_src_height;
#endif /* USE_TEXTURE_MEMORY */

  // Use the most efficient vectorized version
  for(unsigned int v=4;v>0;v=v/2) {
    if(img_dst_width % v == 0) {
      const unsigned int img_dst_width_n = img_dst_width/v;
      const unsigned int thread_num_x = min(64,((img_dst_width_n+32-1)/32)*32);
      const kernel_params_t p = cuda_gen_kernel_params_2d(img_dst_width_n,img_dst_height,thread_num_x,2);
      switch(v) {
        case 1:
#if USE_TEXTURE_MEMORY
          image_halfsample_gpu_tex_kernel<uchar1,1><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar1*)d_img_dst,
            #if USE_TEXTURE_OBJECTS
            tex_object,
            #endif
            img_dst_width,
            img_dst_height,
            img_dst_pitch
          );
#else
          image_halfsample_gpu_kernel<uchar1,1><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar2*)d_img_src,
            (unsigned int)img_src_pitch,
            (uchar1*)d_img_dst,
            (unsigned int)img_dst_width,
            (unsigned int)img_dst_height,
            (unsigned int)img_dst_pitch
          );
#endif /* USE_TEXTURE_MEMORY */
          break;
        case 2:
#if USE_TEXTURE_MEMORY
          image_halfsample_gpu_tex_kernel<uchar2,2><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar2*)d_img_dst,
            #if USE_TEXTURE_OBJECTS
            tex_object,
            #endif
            img_dst_width,
            img_dst_height,
            img_dst_pitch
          );
#else
          image_halfsample_gpu_kernel<uchar2,2><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar2*)d_img_src,
            (unsigned int)img_src_pitch,
            (uchar2*)d_img_dst,
            (unsigned int)(img_dst_width_n),
            (unsigned int)(img_dst_height),
            (unsigned int)(img_dst_pitch)
          );
#endif /* USE_TEXTURE_MEMORY */
          break;
        case 4:
#if USE_TEXTURE_MEMORY
          image_halfsample_gpu_tex_kernel<uchar4,4><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar4*)d_img_dst,
            #if USE_TEXTURE_OBJECTS
            tex_object,
            #endif
            img_dst_width,
            img_dst_height,
            img_dst_pitch
          );
#else
          image_halfsample_gpu_kernel<uchar4,4><<<p.blocks_per_grid,p.threads_per_block,0,stream>>>(
            (uchar2*)d_img_src,
            (unsigned int)img_src_pitch,
            (uchar4*)d_img_dst,
            (unsigned int)(img_dst_width_n),
            (unsigned int)(img_dst_height),
            (unsigned int)(img_dst_pitch)
          );
#endif /* USE_TEXTURE_MEMORY */
          break;
      }
      break;
    }
  }

#if USE_TEXTURE_MEMORY
#if USE_TEXTURE_OBJECTS
  // Destroy the created texture object
  CUDA_API_CALL(hipDestroyTextureObject(tex_object));
#else
  // Unbind the texture memory
  CUDA_API_CALL(hipUnbindTexture(d_image_in_tex));
#endif /* USE_TEXTURE_OBJECTS */
#endif /* USE_TEXTURE_MEMORY */
}

__host__ void pyramid_create_gpu(std::vector<unsigned char *> & d_images,
                                 std::vector<std::size_t> & width,
                                 std::vector<std::size_t> & height,
                                 std::vector<std::size_t> & pitch,
                                 unsigned int levels,
                                 hipStream_t stream) {
  for(std::size_t l=1;l<levels;++l) {
    pyramid_create_level_gpu(d_images[l-1],
                             d_images[l],
                             pitch[l-1],
                             pitch[l],
                             width[l-1],
                             width[l],
                             height[l-1],
                             height[l],
                             stream);
  }
}

__host__ void pyramid_create_gpu(std::vector<std::shared_ptr<Subframe>> & d_subframes,
                                 hipStream_t stream) {
  for(std::size_t l=1;l<d_subframes.size();++l) {
    pyramid_create_level_gpu(d_subframes[l-1]->data_,
                             d_subframes[l]->data_,
                             d_subframes[l-1]->pitch_,
                             d_subframes[l]->pitch_,
                             d_subframes[l-1]->cols,
                             d_subframes[l]->cols,
                             d_subframes[l-1]->rows,
                             d_subframes[l]->rows,
                             stream);
  }
}

__host__ void pyramid_display(const std::vector<std::shared_ptr<Subframe>> & subframes) {
  for(std::size_t l=0;l<subframes.size();++l) {
    subframes[l]->display();
  }
}

} // namespace vilib
